#include "hip/hip_runtime.h"
// gpu_matrix_multiply.cu
#include "csr_matrix.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void spmv_csr_vector_kernel(int m, const int *rowPtr, const int *colInd, const float *values, const float *x, float *y) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < m) {
        float dot = 0.0f;
        for (int idx = rowPtr[row]; idx < rowPtr[row + 1]; ++idx) {
            dot += values[idx] * x[colInd[idx]];
        }
        y[row] += dot;
    }
}

float gpuMatrixMultiply(const CSRMatrix& A, const CSRMatrix& B, float* C) {

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing GPU code
    hipEventRecord(start);

    int *d_rowPtr_A, *d_colInd_A, *d_rowPtr_B, *d_colInd_B;
    float *d_values_A, *d_values_B, *d_C;

    hipMalloc((void**)&d_rowPtr_A, A.rowPtr.size() * sizeof(int));
    hipMalloc((void**)&d_colInd_A, A.colInd.size() * sizeof(int));
    hipMalloc((void**)&d_values_A, A.values.size() * sizeof(float));
    hipMalloc((void**)&d_rowPtr_B, B.rowPtr.size() * sizeof(int));
    hipMalloc((void**)&d_colInd_B, B.colInd.size() * sizeof(int));
    hipMalloc((void**)&d_values_B, B.values.size() * sizeof(float));
    hipMalloc((void**)&d_C, A.rows * B.cols * sizeof(float));

    hipMemcpy(d_rowPtr_A, A.rowPtr.data(), A.rowPtr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colInd_A, A.colInd.data(), A.colInd.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values_A, A.values.data(), A.values.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtr_B, B.rowPtr.data(), B.rowPtr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colInd_B, B.colInd.data(), B.colInd.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values_B, B.values.data(), B.values.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, A.rows * B.cols * sizeof(float));

    dim3 blockDim(1024);
    dim3 gridDim((A.rows + blockDim.x - 1) / blockDim.x);

    spmv_csr_vector_kernel<<<gridDim, blockDim>>>(A.rows, d_rowPtr_A, d_colInd_A, d_values_A, d_values_B, d_C);

    hipMemcpy(C, d_C, A.rows * B.cols * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_rowPtr_A);
    hipFree(d_colInd_A);
    hipFree(d_values_A);
    hipFree(d_rowPtr_B);
    hipFree(d_colInd_B);
    hipFree(d_values_B);
    hipFree(d_C);

    // Stop timing GPU code
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "CUDA kernel execution time: " << elapsedTime / 1000 << " sec\n";
    return elapsedTime/1000;

}
